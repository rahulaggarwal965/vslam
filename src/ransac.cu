#include "hip/hip_runtime.h"
#include "vslam_internal.h"
#include <vector>

struct model {

};

template <typename T, typename M, int block_size = >
__global__
static void ransac(int *ransac_sets, int set_size, T *data, M *model) {

    const int bx = blockIdx.x;
    __shared__ f32 residuals[blockDim.x]:
    auto set = ransac_sets + threadIdx.x * set_size;
    typename M::out r = model->compute_model(set);

    f32 residual = FLT_MAX;
    residuals[threadIdx.x] = residual = model->compute_residual(M::out, set);

    for (int i = blockDim.x / 2; i > 16; i >>= 1) {
        if (i < blockDim.x / 2) {
            residuals[]
        }
    }

} 
